
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

using namespace std;

int rand();
float RandomNumber(float Min, float Max)
{
    return ((float(rand()) / float(RAND_MAX)) * (Max - Min)) + Min;
}


float sigmoid(float x){
    return 1 / (1 + exp(-x));
}

float sigmoid_der(float x){
    return sigmoid(x) * (1 - sigmoid(x));
}

float * dot_matrix(float m1[3], float m2[3]){
    // dot product code will be here
    // m = matrik indeks 1
    static float C[1];
    
    C[0] = 0;
    for (int j = 0; j < 3; j++){
        C[0] +=  m1[j] * m2[j];
    }
    return C;
}

int main(){

    cout << "Neural Network Start" << endl;

    float feature_set[5][3] = {{0,1,0},{0,0,1},{1,0,0},{1,1,0},{1,1,1}};
    float label[5][1] = {{1},{0},{0},{1},{1}};

    float *inputs;
    float suminput;
    float activation1;
    int ri;
    float error, dcost_dpred, dpred_dz, z_delta;

    float weight[3];
    float bias[1][1];
    float learning_rate = 0.005;

    // filling weight with random number
    for(int i = 0; i < 3; i++){
        weight[i] = RandomNumber(-1, 1);
    }
    // Training Phase
    int epoch = 1000;
    for(int i = 0; i < epoch; i++){
        ri = rand() % 5;

        inputs = dot_matrix(feature_set[ri], weight);
        for(int j = 0; j < 5; j++)
            suminput += inputs[j];
        // suminput += bias[0][0];

        activation1 = sigmoid(suminput);
        error = activation1 - label[ri][0];

        dcost_dpred = error;
        dpred_dz = sigmoid_der(activation1);
        z_delta = dcost_dpred * dpred_dz;

        for(int j = 0; j < 3; j++){
            weight[j] -= (learning_rate * inputs[j] * z_delta);
        }
        // bias -= learning_rate * z_delta;

        cout << i <<" activation : " << activation1 << " Data Ke : " << ri << " error : " << error << endl;
    }
    
    return 0; 
}